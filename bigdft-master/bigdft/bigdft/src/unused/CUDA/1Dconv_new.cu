#include "hip/hip_runtime.h"
/****u* CUDA/1Dconv_new.cu
**
** AUTHOR
**  Luigi Genovese
**
** SOURCE
*/

#include <stdio.h>
#include <pthread.h>
#include <semaphore.h>
#include <sched.h>

#define max(a,b) (a > b ? a : b)
#define min(a,b) (a < b ? a : b)

//maximum size of the shared memory array
//conceived for maximize occupancy on a hardware of compute
//capability 1.2 and higher (1024 threads at same time on a given multiprocessor)
#define MAX_SHARED_SIZE 3072 //16*256 4 kB (should be =~ 3.9 kB, try also 3072)
#define HALF_WARP_SIZE 16 // for all architectures
#define NUM_LINES 16 
#define HW_ELEM 1 //this is HALF_WARP_SIZE/NUM_LINES

//parameter related to the Magic Filter convolution
//lowfil + lupfil + 1  must be a multiple of 16
#define LOWFIL 8
#define LUPFIL 7

//convolution filters
#define MFIL0   8.4334247333529341094733325815816e-7f
#define MFIL1  -0.1290557201342060969516786758559028e-4f
#define MFIL2   0.8762984476210559564689161894116397e-4f
#define MFIL3  -0.30158038132690463167163703826169879e-3f
#define MFIL4   0.174723713672993903449447812749852942e-2f
#define MFIL5  -0.942047030201080385922711540948195075e-2f
#define MFIL6   0.2373821463724942397566389712597274535e-1f
#define MFIL7   0.612625895831207982195380597e-1f
#define MFIL8   0.9940415697834003993178616713f
#define MFIL9  -0.604895289196983516002834636e-1f
#define MFIL10 -0.2103025160930381434955489412839065067e-1f
#define MFIL11  0.1337263414854794752733423467013220997e-1f
#define MFIL12 -0.344128144493493857280881509686821861e-2f
#define MFIL13  0.49443227688689919192282259476750972e-3f
#define MFIL14 -0.5185986881173432922848639136911487e-4f
#define MFIL15  2.72734492911979659657715313017228e-6f

//error handling in CUDA
#define CUERR { hipError_t err; \
 if ((err = hipGetLastError()) != hipSuccess) { \
 printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); }}


typedef struct  _par
{
  //unsigned int LinesPerBlock;
  unsigned int ElementsPerBlock;
  //unsigned int ElementsPerHalfWarp;

  int lowfil, lupfil; //structure of f_fct
  int thline[HALF_WARP_SIZE]; //line considered by a thread within the half-warp
  int thelem[HALF_WARP_SIZE]; //elements considered by a thread within the half-warp
  int hwelem_calc[16]; //maximum number of half warps
  int hwelem_copy[16]; //maximum number of half-warps
  int hwoffset_calc[16]; //maximum number of half warps
  int hwoffset_copy[16]; //maximum number of half-warps
  
  //float fil[16];

} par_t;

__constant__ par_t par;

//declare the texture for binding the input psi
texture<float,2, hipReadModeElementType> psi_tex;

#include "kernel_tex.cu"

int dogenconv(int ndat,
	      int n, 
	      float *GPU_idata,
	      float *GPU_odata,
	      int lowfil,
	      int lupfil);

int dogenconv_n(int ndat,
	      int n, 
	      float *GPU_idata,
	      float *GPU_odata,
	      int lowfil,
	      int lupfil);

void correctSequence(int thds,int elem,int * tab);

//create the parameters to be used for calculating the convolution
//with a given stride
void constantParameters(par_t* par,
			unsigned int* num_halfwarps,
			//unsigned int num_lines,
			int n,
			int ndat,
			int lowfil, //lowfil + lupfil must be a multiple of 16
			int lupfil,
			unsigned int* linecuts,
			unsigned int* num_blocks)

{

  //number of lines treated by each block
  //par->LinesPerBlock = num_lines;

  //number of total allowed elements of a input line
  unsigned int num_elem_tot=MAX_SHARED_SIZE/sizeof(float)/NUM_LINES; //between1024and64
  //unsigned int num_elem_tot=MAX_SHARED_SIZE/sizeof(float)/num_lines; //between 1024 and 64
  
  //number of elements of the output
  unsigned int num_elem_max=min(num_elem_tot-lowfil-lupfil-1,n); //between 1008 and 48 for 16-fil

  //number of elements treated by the single half-warp
  //par -> ElementsPerHalfWarp = HALF_WARP_SIZE/num_lines; //it is assumed they are multiples

  //number of pieces in which a line is divided
  //if the line is too small and not a multiple of ElementsPerHalfWarp
  //divide the line in two
  *linecuts=
    //(n <= num_elem_max && n % par -> ElementsPerHalfWarp !=0 ? 2 : (n-1)/num_elem_max+1);
    (n <= num_elem_max && n % HW_ELEM !=0 ? 2 : (n-1)/num_elem_max+1);

  //number of blocks in ndat direction
  //*num_blocks=((ndat-1)/num_lines + 1);
  *num_blocks=((ndat-1)/NUM_LINES + 1);

  //printf("num_elem_tot %i,num_elem_max %i,linecuts %i,num_blocks %i,elemperHW %i \n",
  //num_elem_tot,num_elem_max,*linecuts,*num_blocks, par -> ElementsPerHalfWarp);

  //number of elements treated by each block 
  //this may pose problems for values of n dimensions less than 48
  //when n is not a multiple of ElementsPerHalfWarp
  par->ElementsPerBlock = 
    //min(par->ElementsPerHalfWarp*(((n-1)/(*linecuts))/par->ElementsPerHalfWarp+1),n);
    min(HW_ELEM*(((n-1)/(*linecuts))/HW_ELEM+1),n);

  int halfwarps=16;
  //calculate the maximum number of halfwarps (between 4 and 16)
  for(int i =3; i>=0; --i)
    {
      //if(par->ElementsPerBlock/par->ElementsPerHalfWarp >= 1 << i)
      if(par->ElementsPerBlock/HW_ELEM >= 1 << i)
	{
	  halfwarps = 1 << i;
	  break;
	}
    }

  *num_halfwarps = halfwarps;

  for(int j=0;j < HALF_WARP_SIZE ; ++j)
    {
      //par->thline[j]= j & (num_lines - 1); //num_lines always a power of two 
      //par->thelem[j]= j / num_lines; 

      par->thline[j]= j & (NUM_LINES - 1); //num_lines always a power of two 
      par->thelem[j]= j / NUM_LINES; 
    }

  //define the sequences of the number of elements
  correctSequence(halfwarps,par->ElementsPerBlock/HW_ELEM,par->hwelem_calc);

  correctSequence(halfwarps,(par->ElementsPerBlock+lowfil+lupfil+1)/HW_ELEM,
		  par->hwelem_copy);

  //correctSequence(halfwarps,par->ElementsPerBlock/par->ElementsPerHalfWarp,
  //par->hwelem_calc);

  //correctSequence(halfwarps,(par->ElementsPerBlock+lowfil+lupfil+1)/par->ElementsPerHalfWarp,
  //par->hwelem_copy);


  //define the offsets
  for(int j=0,pos_calc=0,pos_copy=0;j < halfwarps ; ++j)
    {
      par->hwoffset_calc[j]=pos_calc;
      par->hwoffset_copy[j]=pos_copy;
      pos_calc+=HW_ELEM*par->hwelem_calc[j];
      pos_copy+=HW_ELEM*par->hwelem_copy[j];
      //pos_calc+=par->ElementsPerHalfWarp*par->hwelem_calc[j];
      //pos_copy+=par->ElementsPerHalfWarp*par->hwelem_copy[j];

    }
 
  //lowfil and lupfil parameters
  par->lowfil = lowfil;
  par->lupfil = lupfil;

  //printf("ElementsPerBlock %i,HalfWarpCalculatedElements %i,HalfWarpCopiedElements %i,LastHalfWarpCalcElements %i, LastHalfWarpCopiedElements %i \n",
  //par->ElementsPerBlock,par->hwelem_calc[0],par->hwelem_copy[0],
  //par->hwelem_calc[halfwarps-1],par->hwelem_copy[halfwarps-1]);

  /*
   //filter values for this convolution, hard coded
  par->fil[0] = 8.4334247333529341094733325815816e-7f;
  par->fil[1] =-0.1290557201342060969516786758559028e-4f;
  par->fil[2] = 0.8762984476210559564689161894116397e-4f;
  par->fil[3] =-0.30158038132690463167163703826169879e-3f;
  par->fil[4] = 0.174723713672993903449447812749852942e-2f;
  par->fil[5] =-0.942047030201080385922711540948195075e-2f;
  par->fil[6] = 0.2373821463724942397566389712597274535e-1f;
  par->fil[7] = 0.612625895831207982195380597e-1f;
  par->fil[8] = 0.9940415697834003993178616713f;
  par->fil[9] =-0.604895289196983516002834636e-1f;
  par->fil[10]=-0.2103025160930381434955489412839065067e-1f;
  par->fil[11]= 0.1337263414854794752733423467013220997e-1f;
  par->fil[12]=-0.344128144493493857280881509686821861e-2f;
  par->fil[13]= 0.49443227688689919192282259476750972e-3f;
  par->fil[14]=-0.5185986881173432922848639136911487e-4f;
  par->fil[15]= 2.72734492911979659657715313017228e-6f;
  */

}

/* //already defined in locpot
//uniformise the tabular of the number of elements treated by each
//thread (counterpart of uniformiseTab)
void correctSequence(int thds,int elem,int * tab)
{
  //put to zero all the values;
  for(int j=0;j< elem; ++j)
    {
      tab[j]=0;
    }

  //then start to fill consecutively until reaching of the end
  //if elem > thds no element will be zero
  //this is the most balanced choice
  for(int i=0;i< elem; ++i)
    {
      tab[i % thds]+=1;
    }
}
*/


//1D convolution of multiple lines in the same block
__global__ void conv1d_stride(int n,int ndat, float *psi_in, float *psi_out)
{

  //line treated by the given block
  unsigned int lineOffset = min(blockIdx.y*NUM_LINES,ndat-NUM_LINES);
  //starting element treated by the block
  unsigned int elemOffset = min(blockIdx.x*par.ElementsPerBlock,n-par.ElementsPerBlock);

  //line treated by the given block
  //unsigned int lineOffset = min(blockIdx.y*par.LinesPerBlock,ndat-par.LinesPerBlock);
  //starting element treated by the block
  //unsigned int elemOffset = min(blockIdx.x*par.ElementsPerBlock,n-par.ElementsPerBlock);

  //half-warp id
  const unsigned int hwid = threadIdx.y;
  //tid within the HW
  const unsigned int tid_hw = threadIdx.x;

  //shared memory array
  __shared__ float psi_sh[MAX_SHARED_SIZE/sizeof(float)];

  //line treated by the given thread in ndat axis
  //which is the input base element
  unsigned int BaseElem = par.thline[tid_hw] + lineOffset;
  //write data in shared memory
  //element treated by the given thread in n-axis
  unsigned int thelem = par.thelem[tid_hw] + par.hwoffset_copy[hwid];

  unsigned int ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_copy[hwid];
  //unsigned int ShBaseElem = tid_hw + par.LinesPerBlock*par.hwoffset_copy[hwid];

  int epsilon,npos;

  //NOTE: it is assumed that for non-first segments the starting
  //points is far enough for the filter to be contained
  //and the same for non-last segments.
  //in other terms: lenght of the line is always bigger than
  //max(lowfil,lupfil)

  for(int i=0,ipos=elemOffset-par.lowfil+thelem;i < par.hwelem_copy[hwid] ; ++i)
    {
      epsilon=(ipos < 0 ? -1 : ipos/n);
      npos=ipos-epsilon*n;
      psi_sh[ShBaseElem]=psi_in[BaseElem+ndat*npos];
      //psi_sh[ShBaseElem]=tex1Dfetch(psi_tex,BaseElem+ndat*npos);

      ShBaseElem += HALF_WARP_SIZE;
      ipos += HW_ELEM;
      //ipos += par.ElementsPerHalfWarp;
      
    }

  //end shared memory copy
  __syncthreads();

  //element treated by the given thread in n-axis
  thelem = par.thelem[tid_hw] + par.hwoffset_calc[hwid];
  //base element for the given thread in shared memory
  ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_calc[hwid];
  //ShBaseElem = tid_hw + par.LinesPerBlock*par.hwoffset_calc[hwid];

  //output base element, from the input one
  BaseElem =  n*BaseElem+ thelem + elemOffset;

  //perform convolution in shared memory 
  //each thread calculate a number of elements, identical for each
  //half-warp
  //#pragma unroll 5 (to be tested if it is important)

  for(int i=0;i < par.hwelem_calc[hwid]; ++i)
    {
      //values of the convolution
      register float conv = 
	//hand-unrolled loop (16 elements for this filter)
	//order changed for increasing the precision

	MFIL0 *psi_sh[ShBaseElem               ] +
	MFIL15*psi_sh[ShBaseElem + 15*NUM_LINES] +
	MFIL1 *psi_sh[ShBaseElem +   NUM_LINES ] +
	MFIL14*psi_sh[ShBaseElem + 14*NUM_LINES] +
	MFIL2 *psi_sh[ShBaseElem + 2*NUM_LINES ] +
	MFIL13*psi_sh[ShBaseElem + 13*NUM_LINES] +
	MFIL3 *psi_sh[ShBaseElem + 3*NUM_LINES ] +
	MFIL12*psi_sh[ShBaseElem + 12*NUM_LINES] +
	MFIL4 *psi_sh[ShBaseElem + 4*NUM_LINES ] +
	MFIL11*psi_sh[ShBaseElem + 11*NUM_LINES] +
	MFIL5 *psi_sh[ShBaseElem + 5*NUM_LINES ] +
	MFIL10*psi_sh[ShBaseElem + 10*NUM_LINES] +
	MFIL6 *psi_sh[ShBaseElem + 6*NUM_LINES ] +
	MFIL9 *psi_sh[ShBaseElem + 9*NUM_LINES ] +
	MFIL7 *psi_sh[ShBaseElem + 7*NUM_LINES ] +
	MFIL8 *psi_sh[ShBaseElem + 8*NUM_LINES ] ;

	/*
	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[15]*psi_sh[ShBaseElem + 15*NUM_LINES] +
	par.fil[1]*psi_sh[ShBaseElem +   NUM_LINES ] +
	par.fil[14]*psi_sh[ShBaseElem + 14*NUM_LINES] +
	par.fil[2]*psi_sh[ShBaseElem + 2*NUM_LINES ] +
	par.fil[13]*psi_sh[ShBaseElem + 13*NUM_LINES] +
	par.fil[3]*psi_sh[ShBaseElem + 3*NUM_LINES ] +
	par.fil[12]*psi_sh[ShBaseElem + 12*NUM_LINES] +
	par.fil[4]*psi_sh[ShBaseElem + 4*NUM_LINES ] +
	par.fil[11]*psi_sh[ShBaseElem + 11*NUM_LINES] +
	par.fil[5]*psi_sh[ShBaseElem + 5*NUM_LINES ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*NUM_LINES] +
	par.fil[6]*psi_sh[ShBaseElem + 6*NUM_LINES ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*NUM_LINES ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*NUM_LINES ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*NUM_LINES ] ;

      
	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[15]*psi_sh[ShBaseElem + 15*par.LinesPerBlock] +
	par.fil[1]*psi_sh[ShBaseElem +   par.LinesPerBlock ] +
	par.fil[14]*psi_sh[ShBaseElem + 14*par.LinesPerBlock] +
	par.fil[2]*psi_sh[ShBaseElem + 2*par.LinesPerBlock ] +
	par.fil[13]*psi_sh[ShBaseElem + 13*par.LinesPerBlock] +
	par.fil[3]*psi_sh[ShBaseElem + 3*par.LinesPerBlock ] +
	par.fil[12]*psi_sh[ShBaseElem + 12*par.LinesPerBlock] +
	par.fil[4]*psi_sh[ShBaseElem + 4*par.LinesPerBlock ] +
	par.fil[11]*psi_sh[ShBaseElem + 11*par.LinesPerBlock] +
	par.fil[5]*psi_sh[ShBaseElem + 5*par.LinesPerBlock ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*par.LinesPerBlock] +
	par.fil[6]*psi_sh[ShBaseElem + 6*par.LinesPerBlock ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*par.LinesPerBlock ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*par.LinesPerBlock ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*par.LinesPerBlock ] ;


	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[1]*psi_sh[ShBaseElem +   par.LinesPerBlock ] +
	par.fil[2]*psi_sh[ShBaseElem + 2*par.LinesPerBlock ] +
	par.fil[3]*psi_sh[ShBaseElem + 3*par.LinesPerBlock ] +
	par.fil[4]*psi_sh[ShBaseElem + 4*par.LinesPerBlock ] +
	par.fil[5]*psi_sh[ShBaseElem + 5*par.LinesPerBlock ] +
	par.fil[6]*psi_sh[ShBaseElem + 6*par.LinesPerBlock ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*par.LinesPerBlock ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*par.LinesPerBlock ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*par.LinesPerBlock ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*par.LinesPerBlock] +
	par.fil[11]*psi_sh[ShBaseElem + 11*par.LinesPerBlock] +
	par.fil[12]*psi_sh[ShBaseElem + 12*par.LinesPerBlock] +
	par.fil[13]*psi_sh[ShBaseElem + 13*par.LinesPerBlock] +
	par.fil[14]*psi_sh[ShBaseElem + 14*par.LinesPerBlock] +
	par.fil[15]*psi_sh[ShBaseElem + 15*par.LinesPerBlock];
      */

      psi_out[BaseElem]=conv;
      //psi_sh[ShBaseElem+par.lowfil*par.LinesPerBlock]; //for testing only

      ShBaseElem += HALF_WARP_SIZE;
      BaseElem += HW_ELEM;
      //BaseElem += par.ElementsPerHalfWarp;

      
    }

 
}


//assume psi_in is a 2d texture
__global__ void conv1d_stride_tex(int n,int ndat,float *psi_out) 
{

  //line treated by the given block
  unsigned int lineOffset = min(blockIdx.y*NUM_LINES,ndat-NUM_LINES);
  //starting element treated by the block
  unsigned int elemOffset = min(blockIdx.x*par.ElementsPerBlock,n-par.ElementsPerBlock);
  //half-warp id
  const unsigned int hwid = threadIdx.y;
  //tid within the HW
  const unsigned int tid_hw = threadIdx.x;

  //shared memory array
  __shared__ float psi_sh[MAX_SHARED_SIZE/sizeof(float)];

  //line treated by the given thread in ndat axis
  //which is the input base element
  unsigned int BaseElem = par.thline[tid_hw] + lineOffset;
  //write data in shared memory
  //element treated by the given thread in n-axis
  //const unsigned int thelem = par.thelem[tid_hw] + par.hwoffset_copy[hwid]
  //unsigned int thelem2 = (unsigned int) par.thelem[tid_hw] + par.hwoffset_copy[hwid];
  unsigned int thelem2 = par.hwoffset_copy[hwid];

  unsigned int ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_copy[hwid];

  //int epsilon,npos;
  //float x,y;

  //NOTE: it is assumed that for non-first segments the starting
  //points is far enough for the filter to be contained
  //and the same for non-last segments.
  //in other terms: lenght of the line is always bigger than
  //max(lowfil,lupfil)

  for(int i=0,ipos=elemOffset-par.lowfil+thelem2;i < par.hwelem_copy[hwid] ; ++i)
    {
      //epsilon=(ipos < 0 ? -1 : ipos/n);
      //npos=ipos-epsilon*n;
      //psi_sh[ShBaseElem]=psi_in[BaseElem+ndat*npos];
      //x=(float) (npos)/;
      //y=(float)npos +0.5f; 
      //x=((float) (BaseElem) + 0.5f )/((float) (ndat));
      //y=((float) (npos) + 0.5f )/((float) (n));
      psi_sh[ShBaseElem]=12.f;//tex2D(psi_tex,x,y);
      //CUERR;
      ShBaseElem += HALF_WARP_SIZE;
      ipos += HW_ELEM;
      
    }

  //end shared memory copy
  __syncthreads();

  //element treated by the given thread in n-axis
  unsigned int thelem = par.thelem[tid_hw] + par.hwoffset_calc[hwid];
  //base element for the given thread in shared memory
  ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_calc[hwid];
  //ShBaseElem = tid_hw + par.LinesPerBlock*par.hwoffset_calc[hwid];

  //output base element, from the input one
  BaseElem =  n*BaseElem + thelem + elemOffset;

  //perform convolution in shared memory 
  //each thread calculate a number of elements, identical for each
  //half-warp
  //#pragma unroll 5 (to be tested if it is important)

  for(int i=0;i < par.hwelem_calc[hwid]; ++i)
    {
    //values of the convolution
    /* register float conv = 
	//hand-unrolled loop (16 elements for this filter)
	//order changed for increasing the precision

	MFIL0 *psi_sh[ShBaseElem               ] +
	MFIL15*psi_sh[ShBaseElem + 15*NUM_LINES] +
	MFIL1 *psi_sh[ShBaseElem +   NUM_LINES ] +
	MFIL14*psi_sh[ShBaseElem + 14*NUM_LINES] +
	MFIL2 *psi_sh[ShBaseElem + 2*NUM_LINES ] +
	MFIL13*psi_sh[ShBaseElem + 13*NUM_LINES] +
	MFIL3 *psi_sh[ShBaseElem + 3*NUM_LINES ] +
	MFIL12*psi_sh[ShBaseElem + 12*NUM_LINES] +
	MFIL4 *psi_sh[ShBaseElem + 4*NUM_LINES ] +
	MFIL11*psi_sh[ShBaseElem + 11*NUM_LINES] +
	MFIL5 *psi_sh[ShBaseElem + 5*NUM_LINES ] +
	MFIL10*psi_sh[ShBaseElem + 10*NUM_LINES] +
	MFIL6 *psi_sh[ShBaseElem + 6*NUM_LINES ] +
	MFIL9 *psi_sh[ShBaseElem + 9*NUM_LINES ] +
	MFIL7 *psi_sh[ShBaseElem + 7*NUM_LINES ] +
	MFIL8 *psi_sh[ShBaseElem + 8*NUM_LINES ] ; */

    psi_out[BaseElem]=//=conv;
	psi_sh[ShBaseElem+par.lowfil*NUM_LINES]; //for testing only

    ShBaseElem += HALF_WARP_SIZE;
    BaseElem += HW_ELEM;
    //BaseElem += par.ElementsPerHalfWarp;

      
    }

 
}


//interface, only the 1d convolution
extern "C" 
void g1dconv_(int *n, 
	      int *ndat, 
	      float **data_in, 
	      float **data_out, 
	      float *filters, 
	      int *lowfil, 
	      int *lupfil)
{

  const int n1 = *ndat;
  const int n2 = *n+1;

  
  if(dogenconv_n(n1,
	       n2, 
	       *data_in,
	       *data_out,
	       *lowfil,
	       *lupfil) != 0)
    {
      return;
    } 
  return; 
}


int dogenconv(int ndat,
	      int n, 
	      float *GPU_idata,
	      float *GPU_odata,
	      int lowfil,
	      int lupfil)
{

  //create the parameters
  par_t parCPU;
  //hipChannelFormatDesc channelDesc = 
  //hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

  hipArray* psiCA;

  //allocate the cuda array for the Psi wavefunction
  //create the channel format descriptor
  //hipChannelFormatDesc channelDesc=
  // hipCreateChannelDesc<float>();

  if(hipMallocArray(&psiCA,&psi_tex.channelDesc,ndat,n) != 0)
    {
      printf("GPU allocation error \n");
      return 1;
    }


  //copy the input array to the cuda Array
  if(hipMemcpyToArray(psiCA,0,0,GPU_idata, ndat*n*sizeof(float),hipMemcpyDeviceToDevice)  != 0)
    {
      printf("DeviceToDevice Memcpy error \n");
      return 1;
    }


  //calculate the number of threads and blocks
  //unsigned int num_lines = min(16,ndat); //hard coded for the moment
  unsigned int numBlocks,linecuts,num_halfwarps;
  //int tex_offset;
  //size_t offset;

  constantParameters(&parCPU,&num_halfwarps,n,ndat,lowfil,lupfil,
		     &linecuts,&numBlocks);
  //constantParameters(&parCPU,&num_halfwarps,num_lines,n,ndat,lowfil,lupfil,
  //&linecuts,&numBlocks);



  //printf("num_blocksx %i, num_blocksy %i, halfwarps %i\n",linecuts,numBlocks,num_halfwarps);

  //send them to constant memory
  if(hipMemcpyToSymbol(HIP_SYMBOL(par),&parCPU, sizeof(par_t)) != 0)
    {
      printf("MemcpyToSymbol error\n");

      return 1;
    }
 
  //define the number of threads and blocks according to parameter definitions
  dim3  grid1(linecuts,  numBlocks, 1);  
  dim3  threads1(HALF_WARP_SIZE, num_halfwarps , 1);

  //set texture parameters
  psi_tex.addressMode[0] = hipAddressModeWrap;
  psi_tex.addressMode[1] = hipAddressModeWrap;
  psi_tex.filterMode = hipFilterModePoint;
  psi_tex.normalized = 1;


  //bind the texture reference to the CUDA array
  hipBindTextureToArray(psi_tex,psiCA,psi_tex.channelDesc);
  CUERR;

  //element offset for reading from the texture
  //tex_offset = offset/sizeof(float);
  
  //printf(" offset %i\n",tex_offset); 
  //launch the kernel grid
  //conv1d_stride_tex <<< grid1, threads1 >>>(n,ndat, GPU_odata);
  //conv1d_stride <<< grid1, threads1 >>>(n,ndat, GPU_idata, GPU_odata);

  conv1d_stride_tex_m<<< grid1, threads1 >>>(n,ndat, GPU_idata, GPU_odata);
  hipDeviceSynchronize();

  //unbind the texture
  hipUnbindTexture(psi_tex);
  CUERR;
  //free the CUDA Array
  //CUDA_SAFE_CALL(hipFreeArray(psiCA));;
  hipFreeArray(psiCA);;

  //hipDeviceSynchronize();

  return 0;

}

int dogenconv_n(int ndat,
	      int n, 
	      float *GPU_idata,
	      float *GPU_odata,
	      int lowfil,
	      int lupfil)
{

  //create the parameters
  par_t parCPU;

  //calculate the number of threads and blocks
  unsigned int numBlocks,linecuts,num_halfwarps;

  constantParameters(&parCPU,&num_halfwarps,n,ndat,lowfil,lupfil,
		     &linecuts,&numBlocks);

  //send them to constant memory
  if(hipMemcpyToSymbol(HIP_SYMBOL(par),&parCPU, sizeof(par_t)) != 0)
    {
      printf("MemcpyToSymbol error\n");

      return 1;
    }
 
  //define the number of threads and blocks according to parameter definitions
  dim3  grid1(linecuts,  numBlocks, 1);  
  dim3  threads1(HALF_WARP_SIZE, num_halfwarps , 1);

  //launch the kernel grid
  conv1d_stride <<< grid1, threads1 >>>(n,ndat, GPU_idata, GPU_odata);
  hipDeviceSynchronize();
  
  return 0;

}

/****/
